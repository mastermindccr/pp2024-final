#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "sys/time.h"

#define CHARSET "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz"
#define CHARSET_SIZE 62
#define THREADS_PER_BLOCK 256
#define batchSize 1000000000

#ifndef PASSWORD_LENGTH
#define PASSWORD_LENGTH 5
#endif

__device__ const uint32_t S[] = {
    7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
    5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,
    4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
    6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21
};

__device__ const uint32_t K[] = {
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be, 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c, 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1, 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

#define F(X, Y, Z) ((X & Y) | (~X & Z))
#define G(X, Y, Z) ((X & Z) | (Y & ~Z))
#define H(X, Y, Z) (X ^ Y ^ Z)
#define I(X, Y, Z) (Y ^ (X | ~Z))

struct MD5Context {
    uint64_t size;
    uint32_t buffer[4]; 
    uint8_t input[64];
    uint8_t digest[16];
};

__device__ uint32_t rotateLeft(uint32_t x, uint32_t n) {
    return (x << n) | (x >> (32 - n));
}

__device__ void md5InitDevice(MD5Context *ctx) {
    ctx->size = (uint64_t)0;
    ctx->buffer[0] = 0x67452301;
    ctx->buffer[1] = 0xefcdab89;
    ctx->buffer[2] = 0x98badcfe;
    ctx->buffer[3] = 0x10325476;
}

__device__ void md5StepDevice(uint32_t *buffer, uint32_t *input){
    uint32_t AA = buffer[0];
    uint32_t BB = buffer[1];
    uint32_t CC = buffer[2];
    uint32_t DD = buffer[3];

    uint32_t E;

    unsigned int j;

    for(unsigned int i = 0; i < 64; ++i){
        switch(i / 16){
            case 0:
                E = F(BB, CC, DD);
                j = i;
                break;
            case 1:
                E = G(BB, CC, DD);
                j = ((i * 5) + 1) % 16;
                break;
            case 2:
                E = H(BB, CC, DD);
                j = ((i * 3) + 5) % 16;
                break;
            default:
                E = I(BB, CC, DD);
                j = (i * 7) % 16;
                break;
        }

        uint32_t temp = DD;
        DD = CC;
        CC = BB;
        BB = BB + rotateLeft(AA + E + K[i] + input[j], S[i]);
        AA = temp;
    }

    buffer[0] += AA;
    buffer[1] += BB;
    buffer[2] += CC;
    buffer[3] += DD;
}

__device__ void md5UpdateDevice(MD5Context *ctx, uint8_t *input_buffer, size_t input_len){
    uint32_t input[16];
    unsigned int offset = ctx->size % 64;
    ctx->size += (uint64_t)input_len;

    // Copy each byte in input_buffer into the next space in our context input
    for(unsigned int i = 0; i < input_len; ++i){
        ctx->input[offset++] = (uint8_t)*(input_buffer + i);

        // If we've filled our context input, copy it into our local array input
        // then reset the offset to 0 and fill in a new buffer.
        // Every time we fill out a chunk, we run it through the algorithm
        // to enable some back and forth between cpu and i/o
        if(offset % 64 == 0){
            for(unsigned int j = 0; j < 16; ++j){
                // Convert to little-endian
                // The local variable `input` our 512-bit chunk separated into 32-bit words
                // we can use in calculations
                input[j] = (uint32_t)(ctx->input[(j * 4) + 3]) << 24 |
                           (uint32_t)(ctx->input[(j * 4) + 2]) << 16 |
                           (uint32_t)(ctx->input[(j * 4) + 1]) <<  8 |
                           (uint32_t)(ctx->input[(j * 4)]);
            }
            md5StepDevice(ctx->buffer, input);
            offset = 0;
        }
    }
}

__device__ void md5FinalizeDevice(MD5Context *ctx){
	uint8_t PADDING[64] = {0};
    PADDING[0] = 0x80;

    uint32_t input[16];
    unsigned int offset = ctx->size % 64;
    unsigned int padding_length = offset < 56 ? 56 - offset : (56 + 64) - offset;

    // Fill in the padding and undo the changes to size that resulted from the update
    md5UpdateDevice(ctx, PADDING, padding_length);
    ctx->size -= (uint64_t)padding_length;

    // Do a final update (internal to this function)
    // Last two 32-bit words are the two halves of the size (converted from bytes to bits)
    for(unsigned int j = 0; j < 14; ++j){
        input[j] = (uint32_t)(ctx->input[(j * 4) + 3]) << 24 |
                   (uint32_t)(ctx->input[(j * 4) + 2]) << 16 |
                   (uint32_t)(ctx->input[(j * 4) + 1]) <<  8 |
                   (uint32_t)(ctx->input[(j * 4)]);
    }
    input[14] = (uint32_t)(ctx->size * 8);
    input[15] = (uint32_t)((ctx->size * 8) >> 32);

    md5StepDevice(ctx->buffer, input);

    // Move the result into digest (convert from little-endian)
    for(unsigned int i = 0; i < 4; ++i){
        ctx->digest[(i * 4) + 0] = (uint8_t)((ctx->buffer[i] & 0x000000FF));
        ctx->digest[(i * 4) + 1] = (uint8_t)((ctx->buffer[i] & 0x0000FF00) >>  8);
        ctx->digest[(i * 4) + 2] = (uint8_t)((ctx->buffer[i] & 0x00FF0000) >> 16);
        ctx->digest[(i * 4) + 3] = (uint8_t)((ctx->buffer[i] & 0xFF000000) >> 24);
    }
}

__device__ void generatePassword(uint64_t idx, char *password, int length) {
    password[length] = '\0';
    for (int i = length - 1; i >= 0; --i) {
        password[i] = CHARSET[idx % CHARSET_SIZE];
        idx /= CHARSET_SIZE;
    }
}

__device__ bool stringCompare(const char *str1, const char *str2) {
    int i = 0;
    while (str1[i] != '\0' && str2[i] != '\0') {
        if (str1[i] != str2[i]) {
            return false;
        }
        i++;
    }
    return str1[i] == '\0' && str2[i] == '\0';
}

__global__ void md5BruteForceKernel(uint8_t *targetHash, uint64_t offset, uint64_t currentBatchSize, bool *d_foundFlag) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (*d_foundFlag || idx >= currentBatchSize + offset) return;

    char candidate[PASSWORD_LENGTH + 1];

    generatePassword(idx, candidate, PASSWORD_LENGTH);

    MD5Context ctx;
    md5InitDevice(&ctx);
    md5UpdateDevice(&ctx, (uint8_t *)candidate, PASSWORD_LENGTH);
    md5FinalizeDevice(&ctx);

    bool match = true;
    for (int i = 0; i < 16; i++) {
        if (ctx.digest[i] != targetHash[i]) {
            match = false;
            break;
        }
    }

    if (match) {
        printf("\nPassword found: %s\n", candidate);
		*d_foundFlag = true;
    }
}

void printProgressBar(double progress) {
	int progressBarWidth = 50;
	printf("\r[");
    int pos = (int)(progress / 100.0 * progressBarWidth);
    for (int i = 0; i < progressBarWidth; i++) {
        if (i < pos)
            printf("=");
        else if (i == pos)
            printf(">");
        else
            printf(" ");
    }
    printf("] %.2f%%", progress);
    fflush(stdout);
}

void md5BruteForceCUDA(const char *targetHashHex) {
	// printf("targetHashHex: %s\n", targetHashHex);
    uint8_t targetHash[16];
    for (int i = 0; i < 16; i++) {
        sscanf(targetHashHex + 2 * i, "%2hhx", &targetHash[i]);
    }
	printf("\n");

	// Allocate memory of hash value
    uint8_t *d_targetHash;
    hipMalloc(&d_targetHash, 16);
    hipMemcpy(d_targetHash, targetHash, 16, hipMemcpyHostToDevice);

	// Allocate memory of flag
	bool h_foundFlag = false; // host variable
    bool *d_foundFlag;        // device variable
	hipMalloc(&d_foundFlag, sizeof(bool));
    hipMemcpy(d_foundFlag, &h_foundFlag, sizeof(bool), hipMemcpyHostToDevice);
	
	// Define and calculate all possible passwords
    uint64_t totalCombinations = pow(CHARSET_SIZE, PASSWORD_LENGTH);
	//totalCombinations = 300000;
    uint64_t remainingCombinations = totalCombinations;
    uint64_t currentOffset = 0;

	double progress = 0.0; // Progress Bar
	while (remainingCombinations > 0) {
        uint64_t currentBatch = (remainingCombinations > batchSize) ? batchSize : remainingCombinations;
        int blocks = (currentBatch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        //printf("Launching kernel with %d blocks, batch size: %lu, offset: %lu\n", blocks, currentBatch, currentOffset);

        md5BruteForceKernel<<<blocks, THREADS_PER_BLOCK>>>(d_targetHash, currentOffset, currentBatch, d_foundFlag);

        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            break;
        }

		// Check password is found
		hipMemcpy(&h_foundFlag, d_foundFlag, sizeof(bool), hipMemcpyDeviceToHost);
		if (h_foundFlag) {
            printf("Early termination: password found.\n");
            break;
        }

		// Update progress
		currentOffset += currentBatch;
		remainingCombinations -= currentBatch;
		progress = (double)currentOffset / totalCombinations * 100.0;
		printProgressBar(progress);
    }
    //cudaDeviceSynchronize();
    hipFree(d_targetHash);
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <MD5 hash>\n", argv[0]);
        return 1;
    }

	struct timeval start, end;
    gettimeofday(&start, NULL);

    md5BruteForceCUDA(argv[1]);

    gettimeofday(&end, NULL);
    double elapsedTime = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;
    printf("\nExecution Time: %f seconds\n", elapsedTime);

    return 0;
}

